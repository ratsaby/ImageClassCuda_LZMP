#include "hip/hip_runtime.h"
/**
  ******************************************************************************
  * @file    gpu_dict.cu
  * @author  alexb
  * @brief   GPU kernels
  ******************************************************************************
  * @attention
  *
  * <h2><center>&copy; Copyright (c) 2015, Alexander Belousov and Joel Ratsaby All rights reserved.</center></h2>
  *
  * LICENSE CONDITIONS
  *
  * Redistribution and use in source and binary forms, with or without modification,
  * are permitted provided that the following conditions are met:
  *   1. Redistributions of source code must retain the above copyright notice,
  *      this list of conditions and the following disclaimer.
  *   2. Redistributions in binary form must reproduce the above copyright notice,
  *      this list of conditions and the following disclaimer in the documentation
  *      and/or other materials provided with the distribution.
  *   3. Neither the name of Ariel University nor the names of its contributors
  *      may be used to endorse or promote products derived from this software
  *      without specific prior written permission.
  *
  * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
  * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
  * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
  * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
  * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
  * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
  * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
  * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
  * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
  *
  ******************************************************************************
  */

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <assert.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> 
#include <img_types.h>
#include <gpu_dict.h>
#include <dist.h>


/**
 *@brief  GPU - kernel 
 *        Calculating LZ-complexity of a single member in a given array, 
 *        Each block works on a calculation of a single member in a given array.
 *        Carefully launch blocks, blocks = number of members in an array. 
 *@param  ptr: device pointer to images pure data array,
 *        This parameter can be a value of @ref img_data_t
 *@param  lzmp_value: device pointer where kernel will copy after calculation  
 *        LZ-complexity of each member in array 
 *@retval None
 *
 *
 * JOEL: This function is what Alex calls gpu_LZMP function in his Thesis book
 *
 */
__device__ void gpu_calc_single_LZMP(char *data, int size, int *lz_val) {
    // initialize Thread index within a block
    int threadId = threadIdx.x;
    __shared__ int m; // history length
    __shared__ int SM; // maximum steps
    int D = 0; // lz variable  (Dictionary)
    int p = MAX_THREADS_PER_BLOCK;
    int n = size;

    int i;
    int k;
    int h;
    int z;
    int j;
    int index;

    m = 0;
    while (m < n) {
        SM = 0;
        if (threadId >= 0 && threadId < p) {
            index = m / p;
            for (int l = 0; l < index + 1; l++) {
                // create new index that depend from threads
                j = threadId + (l * p);
                if (j < m) {
                    i = 0;
                    k = j;
                    h = m - j;
                    // Let each Thread scan and compare characters
                    // in history buffer with characters in S string
                    while (data[m + i] == data[k]) {
                        i++;
                        k++;
                        h--;
                        if (h == 0 || (m + i) == n) {
                            break;
                        }
                    }
                    // If history is over and S is not
                    if (h == 0 && ((m + i) < n)) {
                        z = m;
                        // Let each Thread continue to scan &
                        // compare characters in S string
                        while (data[z] == data[m + i]) {
                            i++;
                            z++;
                            if ((m + i) == n) {
                                break;
                            }
                        }
                    }
                    if (i > 0)
                        atomicMax(&SM, i);
                }
            }
        }
        __syncthreads();
        // Only first thread check and copy to history buffer
        if (threadId == 0) {
            m += SM + 1;
            D++;
        }
        __syncthreads();
    }
    if (threadId == 0)
        *lz_val = D;
}

/**
 *@brief  GPU - kernel 
 *        Calculating LZ-complexity of a single member in a given array, 
 *        Each block works on a calculation of a single member in a given array.
 *        Carefully launch blocks, blocks = number of members in an array. 
 *@param  ptr: device pointer to images pure data array,
 *        This parameter can be a value of @ref img_data_t
 *@param  lzmp_value: device pointer where kernel will copy after calculation  
 *        LZ-complexity of each member in array 
 *@retval None
 *
 *JOEL: this function is what Alex calls gpu_VLZMP in his thesis book
 */
__global__ void gpu_lzmp_single(img_data_t *ptr, int *lzmp_value) {
    // initialize Thread index within a block
    int threadId = threadIdx.x;
    // each block work on specific string
    int member = blockIdx.x;
    int n = ptr[member].len;
    // initialize shared memory array, it much faster than global
    __shared__ char S[MAX_SEQUENCE_ARRAY_SIZE];

    if (threadId == 0) {
        assert(n < MAX_SEQUENCE_ARRAY_SIZE);
        // copy current string from global to shared memory
        memcpy(S, ptr[member].data, n);
    }
    __syncthreads();
    gpu_calc_single_LZMP(S, n, (int*) &lzmp_value[member]);
}

/**
 *@brief  GPU - kernel 
 *        Calculating Distance Matrix of sequence pair arrays,
 *        This kernel will calculate Distance Matrix of sequence pair arrays, 
 *        Each sequence pair calculated in one block.
 *@param  ptr: device pointer to images pure data array,
 *        This parameter can be a value of @ref img_data_t
 *@param  lzmp_value: device pointer to LZ-complexity of each member in array
 *@param  dist_mat: device pointer to Distance Matrix value
 *@param  d_members: device pointer to total members in array
 *@retval None
 */
__global__ void gpu_dist_mat(img_data_t *ptr, int *lzmp_value, float *dist_mat,
        int *d_members) {
    // initialize Thread index within a block
    int threadId = threadIdx.x;
    // each block work on specific string
    int i = blockIdx.x;
    int j = blockIdx.y;
    int n = ptr[i].len + ptr[j].len;
    int lzmp_seq;
    int width = *d_members;
    // initialize shared memory array, it much faster than global
    __shared__ char S[MAX_SEQUENCE_ARRAY_SIZE];

    if (threadId == 0) {
        assert(n < MAX_SEQUENCE_ARRAY_SIZE);
        // copy current string from global to shared memory
        memcpy(S, ptr[i].data, ptr[i].len);
        memcpy(&S[ptr[i].len], ptr[j].data, ptr[j].len);
    }
    __syncthreads();
    gpu_calc_single_LZMP(S, n, &lzmp_seq);
    if (threadId == 0) {
        int index = i + j * width;
        dist_mat[index] = dist_calc_val(lzmp_value[i], lzmp_value[j], lzmp_seq);
    }
}

/**
 *@brief  GPU - kernel 
 *        Calculating Distance Matrix of sequence pair arrays,
 *        This kernel will calculate Distance Matrix of sequence pair arrays, 
 *        Each sequence pair calculated in one block.
 *@param  ptr_row: device pointer to images pure data array per row,
 *        This parameter can be a value of @ref img_data_t
 *@param  lzmp_value_row: device pointer to LZ-complexity of each member in row array
 *@param  row: device pointer to total members in row array
 *@param  ptr_col: device pointer to images pure data array per columns,
 *        This parameter can be a value of @ref img_data_t
 *@param  lzmp_value_col: device pointer to LZ-complexity of each member in column array
 *@param  col: device pointer to total members in column array
 *@param  dist_mat: device pointer to Distance Matrix value
 *@retval None
 */
__global__ void gpu_dist_mat_two_arr(img_data_t *ptr_row, int *lzmp_value_row,
        int *row, img_data_t *ptr_col, int *lzmp_value_col, int *col,
        float* dist_mat) {
    // initialize Thread index within a block
    int threadId = threadIdx.x;
    // each block work on specific string
    int i = blockIdx.x;
    int j = blockIdx.y;
    int n = ptr_row[i].len + ptr_col[j].len;
    int lzmp_seq;
    int width = *col;
    // initialize shared memory array, it much faster than global
    __shared__ char S[MAX_SEQUENCE_ARRAY_SIZE];

    if (threadId == 0) {
        assert(n < MAX_SEQUENCE_ARRAY_SIZE);
        // copy current string from global to shared memory
        memcpy(S, ptr_row[i].data, ptr_row[i].len);
        memcpy(&S[ptr_row[i].len], ptr_col[j].data, ptr_col[j].len);
    }
    __syncthreads();
    gpu_calc_single_LZMP(S, n, &lzmp_seq);
    if (threadId == 0) {
        int index = j + i * width;
        dist_mat[index] = dist_calc_val(lzmp_value_row[i], lzmp_value_col[j], lzmp_seq);
    }
}


/**
 *@brief  Printing Distance Matrix from 1D array as 2D array (Mat).
 *@param  dist_mat: pointer to Distance Matrix 1D array
 *@param  row: split to total rows
 *@param  col: split to total columns
 *@retval None
 */
void gpu_print_dist_mat_1D_array_as_2D_array(float *dist_mat, int row, int col) {
    int index1D;
    printf("Distance Matrix\n");
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            index1D = j + i * col;
            printf("%f ", dist_mat[index1D]);
        }
        printf("\n");
    }
}



/**
 *@brief  Extract images sorted linked list and copy pure data to array.
 *@param  img_list: pointer to linked list of images,
 *        This parameter can be a value of @ref img_struct_t
 *@param  img: pointer to array that data would be copied to,
 *        array should be already allocated,
 *        This parameter can be a value of @ref img_data_t
 *@param  total_img: total images in array
 *@retval None
 */
void gpu_extract_from_list(img_struct_t *img_list, img_data_t *img, int total_img) {
    img_struct_t *ptr_to_free = NULL;
    int currentImg;

    printf("Extracting images data from list - Start ");
    while (img_list != NULL) {
        currentImg = total_img - img_list->img_num - 1; // add to oposite
        assert(img_list->image_size < MAX_ARRAY_SIZE);
        memcpy(img[currentImg].data, img_list->img_data, img_list->image_size);
        img[currentImg].len = img_list->image_size;
        ptr_to_free = img_list;
        img_list = img_list->next;
        free(ptr_to_free);
        printf(".");
    }
    printf("\nExtracting images data from list - Complete\n");
}


/**
 *@brief  Calculating Distance Matrix from single array,
 *		  This function allocate memory on device (GPU)
 *		  Copy data from CPU to GPU and launch the kernels.
 *		  After calculation copy data back and deallocate device memory
 *@param  h_current: host pointer to images pure data array,
 *        This parameter can be a value of @ref img_data_t
 *@param  members: total members in array
 *@param  h_distMat: host pointer to Distance Matrix
 *@retval None
 */
void gpu_calc_dist_mat(img_data_t *h_current, int members, float *h_distMat) {
    img_data_t *d_current;
    int *d_plz_value;
    int *d_members;
    float *d_distMat;

    // Allocate memory on  Device
    checkCudaErrors(hipMalloc((void**) &d_current, members * sizeof (img_data_t) * sizeof (char)));
    checkCudaErrors(hipMalloc((void**) &d_plz_value, members * sizeof (int)));
    checkCudaErrors(hipMalloc((void**) &d_members, sizeof (int)));
    checkCudaErrors(hipMalloc((void**) &d_distMat, members * members * sizeof (float)));
    // Copy data strings from Host memory to Device memory
    checkCudaErrors(hipMemcpy(d_current, h_current, members * sizeof (img_data_t) * sizeof (char),
            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_members, &members, sizeof (int),
            hipMemcpyHostToDevice));
    int dimGrid = members;
    int dimBlock = MAX_THREADS_PER_BLOCK;
    // launch kernel
    gpu_lzmp_single << <dimGrid, dimBlock>>>(d_current, d_plz_value);

    dim3 dimGridSeq(members, members);
    gpu_dist_mat << <dimGridSeq, dimBlock>>>(d_current, d_plz_value, d_distMat, d_members);
    //app_test_kernel<<<dimGrid, dimBlock>>>(d_current, d_plz_value, members);
    hipDeviceSynchronize();
    // Copy result from Device memory to Host memory
    checkCudaErrors(hipMemcpy(h_distMat, d_distMat, members * members * sizeof (float),
            hipMemcpyDeviceToHost));
    // Free Device memory
    checkCudaErrors(hipFree(d_current));
    checkCudaErrors(hipFree(d_plz_value));
    checkCudaErrors(hipFree(d_members));
    checkCudaErrors(hipFree(d_distMat));
}

/**
 *@brief  Calculating Distance Matrix from sequence arrays,
 *		  This function allocate memory on device (GPU)
 *		  Copy data from CPU to GPU and launch the kernels.
 *		  After calculation copy data back and deallocate device memory
 *@param  h_vec_row: host pointer to images pure data array per row,
 *        This parameter can be a value of @ref img_data_t
 *@param  h_row: total members in row array
 *@param  h_vec_col: host pointer to images pure data array per columns,
 *        This parameter can be a value of @ref img_data_t
 *@param  h_col: total members in column array
 *@param  h_distMat: host pointer to Distance Matrix
 *@retval None
 */
void gpu_calc_dist_mat_from_two_arr(img_data_t *h_vec_row, int h_row, img_data_t *h_vec_col, int h_col, float *h_distMat) {
    img_data_t *d_vec_row;
    int *d_row;
    int *d_lzmp_val_row;
    img_data_t *d_vec_col;
    int *d_col;
    int *d_lzmp_val_col;
    float *d_distMat;

    StopWatchInterface *timer = NULL;
    float time_val = 0;
    int total_bytes;

    sdkCreateTimer(&timer);
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    total_bytes = h_row * sizeof (img_data_t) * sizeof (char);
    total_bytes += sizeof (int);
    total_bytes += h_col * sizeof (img_data_t) * sizeof (char);
    total_bytes += sizeof (int);
    printf("Will copy %d bytes to device\n", total_bytes);

    // Allocate memory on  Device
    checkCudaErrors(hipMalloc((void**) &d_vec_row, h_row * sizeof (img_data_t) * sizeof (char)));
    checkCudaErrors(hipMalloc((void**) &d_lzmp_val_row, h_row * sizeof (int)));
    checkCudaErrors(hipMalloc((void**) &d_row, sizeof (int)));

    checkCudaErrors(hipMalloc((void**) &d_vec_col, h_col * sizeof (img_data_t) * sizeof (char)));
    checkCudaErrors(hipMalloc((void**) &d_lzmp_val_col, h_col * sizeof (int)));
    checkCudaErrors(hipMalloc((void**) &d_col, sizeof (int)));

    checkCudaErrors(hipMalloc((void**) &d_distMat, h_row * h_col * sizeof (float)));

    // Copy data strings from Host memory to Device memory
    checkCudaErrors(hipMemcpy(d_vec_row, h_vec_row, h_row * sizeof (img_data_t) * sizeof (char),
            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_row, &h_row, sizeof (int),
            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_vec_col, h_vec_col, h_col * sizeof (img_data_t) * sizeof (char),
            hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_col, &h_col, sizeof (int),
            hipMemcpyHostToDevice));
    sdkStopTimer(&timer);
    time_val = sdkGetTimerValue(&timer);
    printf("Allocate and copy memory to device took:%fms\n", time_val);

    int dimGrid = h_row;
    int dimBlock = MAX_THREADS_PER_BLOCK;
    printf("kernel calc row - start\n");
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);
    // launch kernel
    gpu_lzmp_single << <dimGrid, dimBlock>>>(d_vec_row, d_lzmp_val_row);
    printf("kernel calc row - complete\n");
    printf("kernel calc col - start\n");
    dimGrid = h_col;
    // launch kernel
    gpu_lzmp_single << <dimGrid, dimBlock>>>(d_vec_col, d_lzmp_val_col);
    printf("kernel calc col - complete\n");

    printf("kernel calc dist - start\n");
    dim3 dimGridSeq(h_row, h_col);
    gpu_dist_mat_two_arr << <dimGridSeq, dimBlock>>>(d_vec_row, d_lzmp_val_row, d_row,
            d_vec_col, d_lzmp_val_col, d_col, d_distMat);
    hipDeviceSynchronize();
    sdkStopTimer(&timer);
    time_val = sdkGetTimerValue(&timer);
    sdkDeleteTimer(&timer);
    printf("kernel calc dist - complete\n");
    printf("All kernels took time:%fms\n", time_val);

    // Copy result from Device memory to Host memory
    checkCudaErrors(hipMemcpy(h_distMat, d_distMat, h_row * h_col * sizeof (float),
            hipMemcpyDeviceToHost));
    // Free Device memory
    checkCudaErrors(hipFree(d_vec_row));
    checkCudaErrors(hipFree(d_lzmp_val_row));
    checkCudaErrors(hipFree(d_row));

    checkCudaErrors(hipFree(d_vec_col));
    checkCudaErrors(hipFree(d_lzmp_val_col));
    checkCudaErrors(hipFree(d_col));

    checkCudaErrors(hipFree(d_distMat));
}

/**
 *@brief  Calculating Distance Matrix from single Image sorted linked list,
 *		  This function will extract linked list to an array
 *		  since copy data to GPU available by passing pointer to an array
 *@param  img_list: pointer to images linked list,
 *        This parameter can be a value of @ref img_struct_t
 *@param  dist_mat: pointer to Distance Matrix
 *@param  dist_mat_len: Distance Matrix length
 *@retval true - success , false - not success
 */
bool gpu_dist_mat_from_img_list(img_struct_t *img_list, float *dist_mat, int dist_mat_len) {
    int totalImg;
    int totalImgSeq;
    StopWatchInterface *timer = NULL;
    sdkCreateTimer(&timer);

    printf("Calculation distance matrix from image list - Start\n");
    if (img_list == NULL) {
        printf("Empty image list - Return\n");
        return false;
    }
    totalImg = img_list->img_num + 1;
    totalImgSeq = totalImg * totalImg;
    if (totalImgSeq > dist_mat_len) {
        printf("Distance matrix length is to short - Return\n");
        return false;
    }

    // Should use dynamic memory allocation since our stack is limited.
    img_data_t *img_d = (img_data_t*) malloc(totalImg * sizeof (img_data_t));
    assert(img_d != NULL);

    printf("Total Images:%d, Total Sequences Pair:%d\n", totalImg, totalImgSeq);
    gpu_extract_from_list(img_list, img_d, totalImg);

    printf("Launch GPU kernel - Start\n");
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    gpu_calc_dist_mat(img_d, totalImg, dist_mat);
    sdkStopTimer(&timer);
    printf("Launch GPU kernel - Complete, time:%fms\n",
            sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);
    gpu_print_dist_mat_1D_array_as_2D_array(dist_mat, totalImg, totalImg);

    free(img_d);
    printf("Calculation distance matrix from image list - Complete\n");
    return true;
}

/**
 *@brief  Calculating Distance Matrix from sequence Images sorted linked lists,
 *		  This function will extract linked lists to a sequence arrays
 *		  since copy data to GPU available by passing pointer to an array
 *@param  img_row: pointer to images linked list per rows,
 *        This parameter can be a value of @ref img_struct_t
 *@param  img_col: pointer to images linked list per columns,
 *        This parameter can be a value of @ref img_struct_t
 *@param  dist_mat: pointer to Distance Matrix
 *@param  dist_mat_len: Distance Matrix length
 *@retval true - success , false - not success
 */
bool gpu_dist_mat_from_two_img_list(img_struct_t *img_row, img_struct_t *img_col, float *dist_mat, int dist_mat_len) {
    int row;
    int col;
    int total_img;
    StopWatchInterface *timer = NULL;

    sdkCreateTimer(&timer);
    printf("Calculation distance matrix from two images list - Start\n");
    if (img_row == NULL || img_col == NULL) {
        printf("Empty image list - Return\n");
        return false;
    }

    row = img_row->img_num + 1;
    col = img_col->img_num + 1;
    total_img = row * col;
    if (total_img > dist_mat_len) {
        printf("Distance matrix length is to short - Return\n");
        return false;
    }

    printf("Image rows:%d Image columns:%d Total Sequences Pair:%d\n", row, col,
            total_img);

    // Should use dynamic memory allocation since our stack is limited.
    img_data_t *img_d_row = (img_data_t*) malloc(row * sizeof (img_data_t));
    assert(img_d_row != NULL);
    img_data_t *img_d_col = (img_data_t*) malloc(col * sizeof (img_data_t));
    assert(img_d_col != NULL);

    gpu_extract_from_list(img_row, img_d_row, row);
    gpu_extract_from_list(img_col, img_d_col, col);

    printf("Launch GPU kernel - Start\n");
    sdkResetTimer(&timer);
    sdkStartTimer(&timer);

    gpu_calc_dist_mat_from_two_arr(img_d_row, row, img_d_col, col, dist_mat);
    sdkStopTimer(&timer);
    printf("Launch GPU kernel - Complete, time:%fms\n",
            sdkGetTimerValue(&timer));
    sdkDeleteTimer(&timer);

    gpu_print_dist_mat_1D_array_as_2D_array(dist_mat, row, col);

    free(img_d_row);
    free(img_d_col);
    printf("Calculation distance matrix from image list - Complete\n");
    return true;
}

